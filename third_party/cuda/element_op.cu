#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

#include <iostream>
#include <string>

#include "timer.h"

using namespace std;
using namespace timer;

class ElementOp {
  public:
    virtual ~ElementOp() {}
    virtual void Calc(int n, const float* x, const float* y, float* z) {}
};

class ElementContext {
  public:
    ElementContext(ElementOp* op):
      op_(op)
    {
    }

    void Calc(int n, const float* x, const float* y, float* z) {
      op_->Calc(n, x, y, z);
    }

  private:
    ElementOp* op_;
};

__global__ void AddKernelPerThread(const float* x, const float* y, float* z) {
  int idx = threadIdx.x;
  z[idx] = x[idx] + y[idx];
}

__global__ void AddKernelPerBlock(const float* x, const float* y, float* z) {
  int idx = blockIdx.x;
  z[idx] = x[idx] + y[idx];
}

class ElementAddOpWithThread: public ElementOp {
  public:
    ~ElementAddOpWithThread() {}

    void Calc(int n, const float* x, const float* y, float* z) {
      // allocate GPU buffers
      float* cuda_x = NULL;
      float* cuda_y = NULL;
      float* cuda_z = NULL;
      hipMalloc((void**)&cuda_x, sizeof(float) * n);
      hipMalloc((void**)&cuda_y, sizeof(float) * n);
      hipMalloc((void**)&cuda_z, sizeof(float) * n);

      // copy host memory to GPU buffers
      hipMemcpy(cuda_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
      hipMemcpy(cuda_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

      AddKernelPerThread<<<1, n>>>(cuda_x, cuda_y, cuda_z);

      // hipDeviceSynchronize waits for the kernel to finish, and returns  
      // any errors encountered during the launch.  
      hipDeviceSynchronize();

      // copy GPU buffers to host memory
      hipMemcpy(z, cuda_z, sizeof(float) * n, hipMemcpyDeviceToHost);

      hipFree(cuda_x);
      hipFree(cuda_y);
      hipFree(cuda_z);
    }
};


class ElementAddOpWithBlock: public ElementOp {
  public:
    ~ElementAddOpWithBlock() {}

    void Calc(int n, const float* x, const float* y, float* z) {
      // allocate GPU buffers
      float* cuda_x = NULL;
      float* cuda_y = NULL;
      float* cuda_z = NULL;
      hipMalloc((void**)&cuda_x, sizeof(float) * n);
      hipMalloc((void**)&cuda_y, sizeof(float) * n);
      hipMalloc((void**)&cuda_z, sizeof(float) * n);

      // copy host memory to GPU buffers
      hipMemcpy(cuda_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
      hipMemcpy(cuda_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

      AddKernelPerBlock<<<n, 1>>>(cuda_x, cuda_y, cuda_z);

      hipStreamSynchronize(0);
      // hipDeviceSynchronize waits for the kernel to finish, and returns  
      // any errors encountered during the launch.  
      hipDeviceSynchronize();

      // copy GPU buffers to host memory
      hipMemcpy(z, cuda_z, sizeof(float) * n, hipMemcpyDeviceToHost);

      hipFree(cuda_x);
      hipFree(cuda_y);
      hipFree(cuda_z);
    }
};

class ElementAddOpWithStream: public ElementOp {
  public:
    ~ElementAddOpWithStream() {}

    void Calc(int n, const float* x, const float* y, float* z) {
      // allocate GPU buffers
      float* cuda_x = NULL;
      float* cuda_y = NULL;
      float* cuda_z = NULL;
      hipMalloc((void**)&cuda_x, sizeof(float) * n);
      hipMalloc((void**)&cuda_y, sizeof(float) * n);
      hipMalloc((void**)&cuda_z, sizeof(float) * n);

      // copy host memory to GPU buffers
      hipMemcpy(cuda_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
      hipMemcpy(cuda_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

      // stream
      const int num_streams = 5;
      hipStream_t stream[num_streams];
      for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&stream[i]);
      }

      for (int i = 0; i < num_streams; i++) {
        AddKernelPerBlock<<<1, 1, 0, stream[i]>>>(x + i, y + i, z + i);
      }

      hipDeviceSynchronize();
      // hipDeviceSynchronize waits for the kernel to finish, and returns  
      // any errors encountered during the launch.  
      hipDeviceSynchronize();

      // copy GPU buffers to host memory
      hipMemcpy(z, cuda_z, sizeof(float) * n, hipMemcpyDeviceToHost);

      hipFree(cuda_x);
      hipFree(cuda_y);
      hipFree(cuda_z);
    }
};

void Print(const float* a, int n) {
  for (int i = 0; i < n; i++) {
    cout << a[i] << " ";
  }
  cout << endl;
}

class GPUAutoTimer {
  public:
    GPUAutoTimer(string str=""):
      str_(str)
    {
      hipEventCreate(&start_); // 创建事件
      hipEventCreate(&stop_);
      hipEventRecord(start_, 0); // 记录开始
    }

    ~GPUAutoTimer() {
      hipEventRecord(stop_, 0); // 记录结束
      hipEventSynchronize(stop_); // 事件同步，等待结束事件之前的设备操作均已完成
      float elapsed_time;
      hipEventElapsedTime(&elapsed_time, start_, stop_); // 计算两个事件之间的时长(单位:ms)
      fprintf(stderr, "%s use %.3fms\n", str_.c_str(), elapsed_time);
      hipEventDestroy(start_);
      hipEventDestroy(stop_);
    }

  private:
    string str_;
    hipEvent_t start_; // 事件对象
    hipEvent_t stop_;
};

int main() {
  const int n = 1024;
  float a[n];
  float b[n];
  float c[n];

  for (int i = 0; i < n; i++) {
    a[i] = random() % 10;
    b[i] = random() % 10;
  }
  
  int count;
  hipGetDeviceCount(&count);
  hipSetDevice(0);

  {
  GPUAutoTimer gpu_auto_timer("GPU");
  AutoTimer auto_timer("CPU");
  ElementAddOpWithThread elt_op_add_with_thread;
  ElementContext elt_context1(&elt_op_add_with_thread);
  elt_context1.Calc(n, a, b, c);
  }

  {
  GPUAutoTimer gpu_auto_timer("GPU");
  AutoTimer auto_timer("CPU");
  ElementAddOpWithBlock elt_op_add_with_block;
  ElementContext elt_context2(&elt_op_add_with_block);
  elt_context2.Calc(n, a, b, c);
  }

  {
  GPUAutoTimer gpu_auto_timer("GPU");
  AutoTimer auto_timer("CPU");
  ElementAddOpWithStream elt_op_add_with_stream;
  ElementContext elt_context3(&elt_op_add_with_stream);
  elt_context3.Calc(n, a, b, c);
  }

  // hipDeviceReset must be called before exiting in order for profiling and  
  // tracing tools such as Nsight and Visual Profiler to show complete traces.  
  hipDeviceReset();

  return 0;
}
